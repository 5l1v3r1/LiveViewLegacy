#include "horizontal_mean_filter.cuh"

horizontal_mean_filter::horizontal_mean_filter(int nWidth, int nHeight)
{
	width = nWidth;
	height = nHeight;
	hipSetDevice(HMF_DEVICE_NUM);
	HANDLE_ERROR(hipMalloc((void**)&picture_device,width*height*sizeof(uint16_t)));
	HANDLE_ERROR(hipMalloc((void**)&result_device,width*sizeof(uint16_t)));


	HANDLE_ERROR(hipHostMalloc((void**)&pic_in_host,width*height*sizeof(uint16_t)));
	HANDLE_ERROR(hipHostMalloc((void**)&result_out_host,width*sizeof(uint16_t)));

	hipStreamCreate(&horizontal_stream);
}


horizontal_mean_filter::~horizontal_mean_filter()
{
	hipSetDevice(HMF_DEVICE_NUM);
		HANDLE_ERROR(hipFree(picture_device));
		HANDLE_ERROR(hipFree(result_device));


		HANDLE_ERROR(hipHostFree(pic_in_host));
		HANDLE_ERROR(hipHostFree(result_out_host));

		hipStreamDestroy(horizontal_stream);
}
horizontal_mean_filter::start_horizontal_mean(uint16_t * pic_in)
{

}
boost::shared_array < float > horizontal_mean_filter::wait_horizontal_mean()
{

}
