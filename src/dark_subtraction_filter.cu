#include "hip/hip_runtime.h"
#include "dark_subtraction_filter.cuh"
//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>

#ifndef BYTES_PER_PIXEL
#define BYTES_PER_PIXEL 2
#endif
//This code largely inspired by http://madsravn.dk/posts/simple-image-processing-with-cuda/


//Kernel code, this runs on the GPU (device)
__global__ void pixel_dark_subtraction_filter(u_char * pic_d, u_char * mask_d, int width, int height)
{
	int offset = blockIdx.x*blockDim.x +threadIdx.x; //This gives us how far we are into the u_char

	if(offset < width*height) //Because we needed an interger grid size, we will have a few threads that don't correspond to a location in the image.
	{
	//Each grayscale depth in the u_char * is represented by adjacent bytes in little endian order.
	// For this filter, we don't need to know where we are in the 2D sense since we are only doing a map operation. For gathers or stencils we will need to work on this.
	uint16_t current_value = pic_d[offset*BYTES_PER_PIXEL] | (pic_d[offset*BYTES_PER_PIXEL+1] << 8);
	uint16_t mask_value = mask_d[offset*BYTES_PER_PIXEL] | (mask_d[offset*BYTES_PER_PIXEL+1] << 8);

	current_value -=mask_value; //We could have horrible overflow problems here... Worth checking
	pic_d[offset*BYTES_PER_PIXEL] =(u_char) current_value; //We want the LSB here
	pic_d[offset*BYTES_PER_PIXEL + 1] =(u_char) (current_value << 8); //We want the MSB here

	}



}
u_char * apply_dark_subtraction_filter(u_char * picture_in, u_char * dark_mask, int width, int height)
{
	int pic_size = width*height*BYTES_PER_PIXEL;

	//u_char * picture_in; //Device (GPU) copy of picture in.
	u_char * picture_out = (u_char * )malloc(pic_size); //Create buffer for CPU memory output

	u_char * picture_device;
	u_char * dark_mask_device;

	hipMalloc( (void **)&picture_device, pic_size);
	hipMalloc( (void **)&dark_mask_device, pic_size);


	hipMemcpy(picture_device, picture_in, pic_size, hipMemcpyHostToDevice);
	hipMemcpy(dark_mask_device, dark_mask, pic_size, hipMemcpyHostToDevice);

	//dim3 blockDims(block_length,block_length,1);
	dim3 blockDims(512,1,1);
	dim3 gridDims(ceil((float)width*height/blockDims.x),1,1);


	pixel_dark_subtraction_filter<<<gridDims,blockDims>>>(picture_device, dark_mask_device, width, height);
	hipMemcpy(picture_out,picture_device,pic_size,hipMemcpyDeviceToHost);

	hipFree(picture_device);
	//memcpy(picture_out,picture_in,pic_size);
	return picture_out;
}


