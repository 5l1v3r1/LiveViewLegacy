#include "hip/hip_runtime.h"
#include "chroma_translate_filter.cuh"
#include "cuda_utils.cuh"
#include <iostream>


//MAKING ASSUMPTIONS BECAUSE CHROMA!

#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))
#define UINT16_MAX = 0xFFFF
__global__ void chroma_filter_kernel(uint16_t * pic_d, uint16_t * pic_out_d)
{
	//int offset = blockIdx.x*blockDim.x +threadIdx.x; //This gives us how far we are into the u_char
	unsigned short col = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned short r = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned short width_eigth = WIDTH/8;
	unsigned short c = col / 8;
	unsigned short i = col % 8;
	//	if(col < WIDTH && row < HEIGHT) //Because we needed an interger grid size, we will have a few threads that don't correspond to a location in the image.
	pic_out_d[c+width_eigth*i + r*WIDTH] = (0xFFFF - pic_d[c*8+i + r*WIDTH]);

}
uint16_t * chroma_translate_filter::apply_chroma_translate_filter(uint16_t * picture_in, uint16_t * picture_out)
{

	HANDLE_ERROR(hipSetDevice(CTF_DEVICE_NUM));
	memcpy(pic_in_host,picture_in, PIC_SIZE); //If we stage ourselves it allows for cuda kernel concurrency
	HANDLE_ERROR(hipMemcpy(picture_device, pic_in_host, PIC_SIZE, hipMemcpyHostToDevice));
	//HANDLE_ERROR(hipMemcpy(picture_device, picture_in, PIC_SIZE, hipMemcpyHostToDevice));


	dim3 blockDims(CTF_BLOCK_SIDE,CTF_BLOCK_SIDE,1);
	dim3 gridDims(WIDTH/CTF_BLOCK_SIDE, HEIGHT/CTF_BLOCK_SIDE,1);

	chroma_filter_kernel<<<gridDims,blockDims,0>>>(picture_device, pic_out_d);
	HANDLE_ERROR(hipMemcpy(picture_out,pic_out_d,PIC_SIZE,hipMemcpyDeviceToHost));
	//HANDLE_ERROR(hipStreamSynchronize(chroma_translate_stream)); //blocks until done
	//HANDLE_ERROR(hipDeviceSynchronize());
	HANDLE_ERROR( hipPeekAtLastError() );


	return picture_out;
}
chroma_translate_filter::chroma_translate_filter()
{
	printf("ctf initialized\n");
	HANDLE_ERROR(hipSetDevice(CTF_DEVICE_NUM));

	HANDLE_ERROR(hipMalloc( (void **)&picture_device, PIC_SIZE));
	HANDLE_ERROR(hipMalloc( (void **)&pic_out_d, PIC_SIZE));
	HANDLE_ERROR(hipHostMalloc((void **) &pic_in_host, PIC_SIZE));

	//std::cout << "done alloc" << std::endl;
}
chroma_translate_filter::~chroma_translate_filter()
{
	HANDLE_ERROR(hipSetDevice(CTF_DEVICE_NUM));
	HANDLE_ERROR(hipFree(picture_device));
	HANDLE_ERROR(hipFree(pic_out_d));
	HANDLE_ERROR(hipHostFree(pic_in_host));
}
