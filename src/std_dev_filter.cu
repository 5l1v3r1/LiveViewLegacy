#include "hip/hip_runtime.h"
#include "std_dev_filter.cuh"
#include "cuda_utils.cuh"
#include <hip/hip_runtime_api.h>
#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))

//This is useful for figuring out how to do caching.
//This number was derived from a "ptxas" error, apparently cuda props lied.
#define SHARED_MEM_PER_BLOCK_GTX590 (0x4000 - 0x10)
#define MAX_N 500

#define THREADS_PER_BLOCK SHARED_MEM_PER_BLOCK_GTX590/(BYTES_PER_PIXEL*MAX_N) //If we want to cache into shared memory, this gives us the maximum number of threads per block should be 24 with truncation
//Kernel code, this runs on the GPU (device) uses shared memory to decrease time


__global__ void std_dev_filter_kernel(uint16_t * pic_d, float * picture_out_device, int width, int height, int gpu_buffer_head, int gpu_buffer_tail, int N)
{
	//__shared__ uint16_t cached_block_data [THREADS_PER_BLOCK*MAX_N]; //Should equal 48000Bytes or 24000 uint_16s
	int offset = blockIdx.x*blockDim.x +threadIdx.x; //This gives us how far we are into the u_char
	uint32_t pic_size = height*width*sizeof(uint16_t); //recalculting this value, integer math is cheaper than I/O
	//Doing allocation outside of if because it reduces forked code size?
	float acc = 0;
	float sum = 0; //Really hoping all of these get put in registers
	float mean = 0;
	uint16_t current_val;
	float std_dev;


	if(offset < width*height*BYTES_PER_PIXEL) //Because we needed an interger grid size, we will have a few threads that don't correspond to a location in the image.
	{

		//Put the device global memory into shared memory (reduces amount of slow memory accesses we need to do), also take the sum for averaging


		//for(int i = 0; i <N; i)
		for(int i = 0; i<N; i++) //Get the sum
		{
			//sum += cached_block_data[i+threadIdx.x*MAX_N];
			sum += pic_d[(offset + ((gpu_buffer_head + i )*width*height*sizeof(uint16_t)) ) % pic_size*MAX_N];
		}

		mean = sum/N;
		for(int i = 0; i<N; i++)
		{
			//acc += pow((cached_block_data[i + threadIdx.x*MAX_N] - mean),2);
			acc += pow(pic_d[(offset + ((gpu_buffer_head + i )*width*height*sizeof(uint16_t)) ) % pic_size*MAX_N] - mean,2);
		}

		std_dev = sqrt(acc/(N-1));

		picture_out_device[offset] = std_dev;
		__syncthreads();
	}



}

std_dev_filter::std_dev_filter(int nWidth, int nHeight)
{

	HANDLE_ERROR(hipSetDevice(STD_DEV_DEVICE_NUM));
	width = nWidth; //Making the assumption that all frames in a frame buffer are the same size
	height = nHeight;
	gpu_buffer_head = 0;
	gpu_buffer_tail = 0;
	picture_out= boost::shared_array < float >(new float[width*height]);
	HANDLE_ERROR(hipStreamCreate(&std_dev_stream));
	//std::cout << "threads per block" << THREADS_PER_BLOCK << std::endl;
	HANDLE_ERROR(hipMalloc( (void **)&pictures_device, width*height*sizeof(uint16_t)*MAX_N)); //Allocate a huge amount of memory on the GPU (N times the size of each frame stored as a u_char)
	HANDLE_ERROR(hipMalloc( (void **)&picture_out_device, width*height*sizeof(float))); //Allocate memory on GPU for reduce target

}
std_dev_filter::~std_dev_filter()
{
	HANDLE_ERROR(hipSetDevice(STD_DEV_DEVICE_NUM));
	//All the memory leaks!
	HANDLE_ERROR(hipFree(pictures_device)); //do not free current picture because it poitns to a location inside pictures_device
	HANDLE_ERROR(hipFree(picture_out_device));
	HANDLE_ERROR(hipStreamDestroy(std_dev_stream));
}
void std_dev_filter::update_GPU_buffer(boost::circular_buffer<boost::shared_ptr <frame> > frame_buffer)
{
	HANDLE_ERROR(hipSetDevice(STD_DEV_DEVICE_NUM));

	if(frame_buffer.size() > 1)
	{
		//Incrementally copies data to device (as each frame comes in it gets copied
		HANDLE_ERROR(hipMemcpyAsync(pictures_device + (gpu_buffer_head*width*height*sizeof(uint16_t)),frame_buffer[1].get()->image_data_ptr,width*height*sizeof(uint16_t),hipMemcpyHostToDevice,std_dev_stream));
		if(++gpu_buffer_head == MAX_N) //Increment and test for ring buffer overflow
			gpu_buffer_head = 0;
		if(frame_buffer.size() == MAX_N) //If the frame buffer has been fully populated
		{
			if(++gpu_buffer_tail == MAX_N) //Increment and test for ring buffer overflow
			{
				gpu_buffer_tail = 0;
			}
		}
	}

}
void std_dev_filter::start_std_dev_filter(int N)
{

	HANDLE_ERROR(hipSetDevice(STD_DEV_DEVICE_NUM));

	if(N < MAX_N) //We can't calculate the std. dev farther back in time then we are keeping track.
	{

		//Create thread for each pixel
		dim3 blockDims(THREADS_PER_BLOCK,1,1);

		//+1 to account for possible integer-division truncation
		dim3 gridDims((width*height/blockDims.x +1),1,1);
		uint16_t kernelN = gpu_buffer_head >= gpu_buffer_tail ? gpu_buffer_head - gpu_buffer_tail : gpu_buffer_head - (-gpu_buffer_tail); //account for gpu_buffer_head wraparound
		//std::cout << " kernelN" << kernelN << std::endl;
		std_dev_filter_kernel <<<gridDims,blockDims,0,std_dev_stream>>> (pictures_device, picture_out_device, width, height, gpu_buffer_head, gpu_buffer_tail, N);
		HANDLE_ERROR( hipPeekAtLastError() );

		HANDLE_ERROR(hipMemcpyAsync(picture_out.get(),picture_out_device,width*height*sizeof(uint16_t),hipMemcpyDeviceToHost,std_dev_stream));
		HANDLE_ERROR( hipPeekAtLastError() );

		//hipProfilerStop();

		//return std_dev_stream;
		//return result;

	}
	else
	{
		//std::cerr << "Couldn't take std. dev, N exceeded length of history or maximum alllowed N (" << MAX_N << ")" << std::endl;
		std::fill_n(picture_out.get(),width*height,-1); //Fill with -1 to indicate fail
	}

}
boost::shared_array <float> std_dev_filter::wait_std_dev_filter()
{
	HANDLE_ERROR(hipSetDevice(STD_DEV_DEVICE_NUM));
	HANDLE_ERROR(hipStreamSynchronize(std_dev_stream)); //blocks until done
	HANDLE_ERROR( hipPeekAtLastError() );

	return picture_out;
}


