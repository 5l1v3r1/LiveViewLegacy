#include "hip/hip_runtime.h"
#include "std_dev_filter.cuh"
#include "cuda_utils.cuh"
#include <hip/hip_runtime_api.h>
#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))




__global__ void std_dev_filter_kernel(uint16_t * pic_d, float * picture_out_device, int width, int height, int gpu_buffer_head, int N)
{
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int offset = col + row*width;

	float sum = 0; //Really hoping all of these get put in registers
	float sq_sum = 0;
	float mean = 0;
	float std_dev;
	int value = 0;

	if(offset == 100*width && DEBUG)
	{
		printf("sum: %f sq_sum: %f \n",sum,sq_sum);
	}
	for(int i = 0; i < N; ++i) {
		//index = (gpu_buffer_head - i) < 0 ? (gpu_buffer_head + MAX_N - i)*width*height*sizeof(uint16_t) + offset : (gpu_buffer_head-i)*width*height*sizeof(uint16_t) + offset;
		//index = (gpu_buffer_head-i)*width*height*sizeof(uint16_t) + offset;
		if((gpu_buffer_head -i) >= 0)
		{
			value = *(pic_d + offset+(width*height*(gpu_buffer_head-i)));
		}
		else
		{
			if(offset == 100*width && DEBUG)
			{
				printf("MAX_N: %i GPU_BUF_HEAD: %i i: %i",MAX_N, gpu_buffer_head, i);
			}
			value = *(pic_d + offset+(width*height*(MAX_N - (i-gpu_buffer_head))));
		}
		sum += value;
		sq_sum += value * value;
		if(offset == 100*width && DEBUG)
		{
			printf("value @ line 100: %i sum: %f sq_sum: %f \n",value,sum,sq_sum);
		}
	}
	mean = sum / N;
	std_dev = sqrt(sq_sum / N - mean * mean);
	if(offset == 100*width && DEBUG)
	{
		printf("mean: %f std_dev: %f @ line 100",mean, std_dev);
	}
	picture_out_device[offset] = std_dev;

}

std_dev_filter::std_dev_filter(int nWidth, int nHeight)
{

	HANDLE_ERROR(hipSetDevice(STD_DEV_DEVICE_NUM));
	width = nWidth; //Making the assumption that all frames in a frame buffer are the same size
	height = nHeight;
	gpu_buffer_head = 0;
	currentN = 0;
	picture_out= boost::shared_array < float >(new float[width*height]);
	HANDLE_ERROR(hipStreamCreate(&std_dev_stream));
	//std::cout << "threads per block" << THREADS_PER_BLOCK << std::endl;
	HANDLE_ERROR(hipMalloc( (void **)&pictures_device, width*height*sizeof(uint16_t)*MAX_N)); //Allocate a huge amount of memory on the GPU (N times the size of each frame stored as a u_char)
	HANDLE_ERROR(hipMalloc( (void **)&picture_out_device, width*height*sizeof(float))); //Allocate memory on GPU for reduce target
	HANDLE_ERROR(hipHostMalloc( (void **)&picture_out_host, width*height*sizeof(float))); //Allocate memory on GPU for reduce target
	HANDLE_ERROR(hipHostMalloc( (void **)&picture_in_host, width*height*sizeof(uint16_t))); //Allocate memory on GPU for reduce target

}
std_dev_filter::~std_dev_filter()
{
	HANDLE_ERROR(hipSetDevice(STD_DEV_DEVICE_NUM));
	HANDLE_ERROR(hipFree(pictures_device)); //do not free current picture because it poitns to a location inside pictures_device
	HANDLE_ERROR(hipFree(picture_out_device));
	HANDLE_ERROR(hipHostFree(picture_out_host));
	HANDLE_ERROR(hipHostFree(picture_in_host));

	HANDLE_ERROR(hipStreamDestroy(std_dev_stream));
}
void std_dev_filter::update_GPU_buffer(uint16_t * image_ptr)
{
	//Synchronous Part
	HANDLE_ERROR(hipSetDevice(STD_DEV_DEVICE_NUM));
	HANDLE_ERROR(hipStreamSynchronize(std_dev_stream)); //Turns out this does need to block :(

	memcpy(picture_in_host, image_ptr,width*height*sizeof(uint16_t));
	char *device_ptr = ((char *)(pictures_device)) + (gpu_buffer_head*width*height*sizeof(uint16_t));

	//Asynchronous Part
	HANDLE_ERROR(hipMemcpyAsync(device_ptr ,picture_in_host,width*height*sizeof(uint16_t),hipMemcpyHostToDevice,std_dev_stream)); 	//Incrementally copies data to device (as each frame comes in it gets copied

	//Synchronous again
	if(++gpu_buffer_head == MAX_N) //Increment and test for ring buffer overflow
		gpu_buffer_head = 0; //If overflow, than start overwriting the front
	if(currentN < MAX_N) //If the frame buffer has not been fully populated
	{
		currentN++; //Increment how much history is available
	}


}
void std_dev_filter::start_std_dev_filter(int N)
{

	HANDLE_ERROR(hipSetDevice(STD_DEV_DEVICE_NUM));
	if(N < MAX_N && N<= currentN) //We can't calculate the std. dev farther back in time then we are keeping track.
	{
		dim3 blockDims(BLOCK_SIDE,BLOCK_SIDE,1);
		dim3 gridDims(width/blockDims.x, height/blockDims.y,1);

		//Asynchronous Part
		std_dev_filter_kernel <<<gridDims,blockDims,0,std_dev_stream>>> (pictures_device, picture_out_device, width, height, gpu_buffer_head, N);
		//HANDLE_ERROR( hipPeekAtLastError() );
		HANDLE_ERROR(hipMemcpyAsync(picture_out_host,picture_out_device,width*height*sizeof(float),hipMemcpyDeviceToHost,std_dev_stream));
		//HANDLE_ERROR( hipPeekAtLastError() );
	}
	else
	{
		std::cerr << "Couldn't take std. dev, N exceeded length of history or maximum alllowed N (" << MAX_N << ")" << std::endl;
		std::fill_n(picture_out.get(),width*height,-1); //Fill with -1 to indicate fail
	}

}
uint16_t * std_dev_filter::getEntireRingBuffer() //For testing only
{
	HANDLE_ERROR(hipSetDevice(STD_DEV_DEVICE_NUM));
	uint16_t * out = new uint16_t[width*height*MAX_N];
	HANDLE_ERROR(hipMemcpy(out,pictures_device,width*height*sizeof(uint16_t)*MAX_N,hipMemcpyDeviceToHost));
	return out;
}
boost::shared_array <float> std_dev_filter::wait_std_dev_filter()
{

	HANDLE_ERROR(hipSetDevice(STD_DEV_DEVICE_NUM));
	HANDLE_ERROR(hipStreamSynchronize(std_dev_stream)); //blocks until done
	HANDLE_ERROR( hipPeekAtLastError() );
	memcpy(picture_out.get(),picture_out_host,width*height*sizeof(float));

	return picture_out;
}


